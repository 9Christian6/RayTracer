#include "hip/hip_runtime.h"
#include "CUDAShape.hpp"
#include "Ray.hpp"
// #include "fEquals.hpp"
#include <limits.h>

namespace raytracer
{
    __host__ __device__ bool equals(double lhs, double rhs)
    {
        return std::abs(lhs - rhs) <= __DBL_EPSILON__;
    }

    __host__ __device__ Vector3 newVector3(double x, double y, double z)
    {
        Vector3 s_vec3;
        s_vec3._x = x;
        s_vec3._y = y;
        s_vec3._z = z;
        return s_vec3;
    }

    __host__ __device__ Image newImage(int width, int height)
    {
        Image img;
        img._width = width;
        img._height = height;
        img._pixels = new Color3[width * height];
        return img;
    }

    // __host__ __device__ Vector3 Vector3_new(const Vector &vec)
    // {
    //     Vector3 s_vec3;
    //     s_vec3._x = vec.x();
    //     s_vec3._y = vec.y();
    //     s_vec3._z = vec.z();
    //     return s_vec3;
    // }

    __host__ __device__ Color3 S_Color_new(double r, double g, double b)
    {
        Color3 c;
        c._r = r;
        c._g = g;
        c._b = b;
        return c;
    }

    // __host__ __device__ Ray3 S_ray_new(const Ray &ray)
    // {
    //     Ray3 r;
    //     r._d = Vector3_new(ray.direction());
    //     r._o = Vector3_new(ray.origin());
    //     return r;
    // }

    __host__ __device__ Ray3 newRay3(Vector3 &origin, Vector3 &direction)
    {
        Ray3 r;
        r._o = origin;
        r._d = direction;
        return r;
    }

    __host__ __device__ Vector3 normalize(const Vector3 &op)
    {
        return (op * (1 / length((op))));
    }

    __host__ __device__ Camera3 newCamera(Vector3 &position, Vector3 &forward, Vector3 &up, double fov, double aspectRatio)
    {
        Camera3 cam;
        cam._pos = position;
        forward = position + forward;
        up = position + up;
        cam._forward = normalize(forward - position);
        cam._right = normalize(crossProduct(cam._forward, up));
        cam._up = crossProduct(cam._right, cam._forward);
        cam._h = std::tan(fov);
        cam._w = cam._h * aspectRatio;
        return cam;
    }

    // __host__ __device__ Intersection3 S_intersection_new()
    // {
    //     Intersection3 intersection;
    //     intersection.hit = false;
    //     return intersection;
    // }

    __host__ __device__ Vector3 operator+(const Vector3 &lhs, const Vector3 &rhs)
    {
        Vector3 sum;
        sum._x = lhs._x + rhs._x;
        sum._y = lhs._y + rhs._y;
        sum._z = lhs._z + rhs._z;
        return sum;
    }

    __host__ __device__ Vector3 operator-(const Vector3 &lhs, const Vector3 &rhs)
    {
        Vector3 diff;
        diff._x = lhs._x - rhs._x;
        diff._y = lhs._y - rhs._y;
        diff._z = lhs._z - rhs._z;
        return diff;
    }

    __host__ __device__ Vector3 operator*(double scale, Vector3 &op)
    {
        Vector3 prod;
        prod._x = op._x * scale;
        prod._y = op._y * scale;
        prod._z = op._z * scale;
        return prod;
    }

    __host__ __device__ Vector3 operator*(const Vector3 &op, double scale)
    {
        Vector3 prod;
        prod._x = op._x * scale;
        prod._y = op._y * scale;
        prod._z = op._z * scale;
        return prod;
    }

    __host__ __device__ double operator*(const Vector3 &lhs, const Vector3 &rhs)
    {
        return lhs._x * rhs._x + lhs._y * rhs._y + lhs._z * rhs._z;
    }

    __host__ __device__ double length(const Vector3 &op)
    {
        return std::sqrt(op * op);
    }

    __host__ __device__ double dotPorduct(const Vector3 &lhs, const Vector3 &rhs)
    {
        return lhs._x * rhs._x + lhs._y * rhs._y + lhs._z * rhs._z;
    }

    __host__ __device__ Vector3 crossProduct(const Vector3 &lhs, const Vector3 &rhs)
    {
        double x, y, z;
        x = lhs._y * rhs._z - lhs._z * rhs._y;
        y = lhs._z * rhs._x - lhs._x * rhs._z;
        z = lhs._x * rhs._y - lhs._y * rhs._x;
        return newVector3(x, y, z);
    }

    __host__ __device__ bool orthogonal(const Vector3 &lhs, const Vector3 &rhs)
    {
        // return (dotPorduct(lhs, rhs) <= __DBL_EPSILON__);
        return raytracer::equals(dotPorduct(lhs, rhs), 0);
    }

    __host__ __device__ bool plane_contains(Plane3 plane, Vector3 point)
    {
        return orthogonal(plane._n, plane._o - point);
    }

    __host__ __device__ double lambert(const Vector3 &light, const Vector3 &position, const Vector3 &normal)
    {
        double brightness{0};
        auto lightDirection = light - position;
        lightDirection = normalize(lightDirection);
        brightness = normal * lightDirection;
        return std::abs(brightness);
    }

    // __host__ __device__ bool visible(thrust::host_vector<TaggedShape> scene, Vector3 position, Vector3 light)
    // {
    //     double t = length(position - light);
    //     auto lightDirection = normalize(position - light);
    //     auto lightRay = S_ray_new(light - 0.1 * lightDirection, lightDirection);
    //     auto firstLightHit = intersectShapes(scene, lightRay).t;
    //     return (t < firstLightHit);
    // }

    __host__ __device__ double calculateLambert(Intersection3 hit, Vector3 light)
    {
        double shade = lambert(light, hit._position, hit._normal);
        return shade;
    }

    // __host__ __device__ double calculateLambert(thrust::host_vector<TaggedShape> shapes, Intersection3 hit, thrust::host_vector<Vector3> lights)
    // {
    //     double shade{0};
    //     for (auto light : lights)
    //     {
    //         if (visible(shapes, hit._position, light))
    //             shade += lambert(light, hit._position, hit._normal);
    //     }
    //     return shade;
    // }

    __host__ __device__ Intersection3 intersectShape(TaggedShape shape, Ray3 ray)
    {
        Intersection3 intersection;
        Plane3 plane;
        Sphere3 sphere;
        double denom{0}, t{0}, B, C;
        bool contains{false};
        Vector3 sum, prod, normal;
        switch (shape._tag)
        {
        case SPHERE:
            sphere = shape._shape._sphere;
            B = ray._d._x * (ray._o._x - sphere._o._x);
            B += ray._d._y * (ray._o._y - sphere._o._y);
            B += ray._d._z * (ray._o._z - sphere._o._z);
            B *= 2;
            C = std::pow((ray._o._x - sphere._o._x), 2);
            C += std::pow((ray._o._y - sphere._o._y), 2);
            C += std::pow((ray._o._z - sphere._o._z), 2);
            C -= std::pow(sphere._r, 2);
            t = std::pow(B, 2) - (4 * C);
            if (t > RAY_T_MIN && t < RAY_T_MAX)
            {
                t = std::sqrt(t);
                t = -B - t;
                if (t <= RAY_T_MIN)
                {
                    break;
                }
                t /= 2;
                intersection.hit = true;
                intersection.t = t;
                intersection._ray = ray;
                prod = t * ray._d;
                sum = ray._o + prod;
                normal = sum - sphere._o;
                intersection.lambert = lambert(ray._o, sum, normal);
                intersection._color = shape._color;
                intersection._normal = normal;
                intersection._position = calculateRayPoint(intersection._ray, intersection.t);
                intersection._shape = SPHERE;
            }
            break;

        case PLANE:
            plane = shape._shape._plane;
            denom = dotPorduct(plane._n, ray._d);
            contains = orthogonal(plane._n, plane._o - ray._o);
            if (equals(denom, 0) && !(plane_contains(plane, ray._o)))
            {
                intersection.hit = false;
                break;
            }
            if (equals(denom, 0) && (plane_contains(plane, ray._o)))
            {
                intersection.hit = true;
                intersection._ray = ray;
                intersection.t = 1;
                intersection._normal = plane._n;
                intersection._shape = PLANE;
                break;
            }
            t = dotPorduct(plane._o - ray._o, plane._n) / denom;
            if (t > 0)
            {
                intersection.hit = true;
                intersection._ray = ray;
                intersection.t = t;
                intersection._position = calculateRayPoint(intersection._ray, t);
                prod = t * ray._d;
                sum = ray._o + prod;
                intersection.lambert = lambert(ray._o, sum, plane._n);
                intersection._normal = plane._n;
                intersection._shape = PLANE;
                break;
            }

        default:
            break;
        }
        return intersection;
    };

    // __host__ __device__ Color3 getPixel(Image img, size_t x, size_t y)
    // {
    //     int position = y * img._width + x;
    //     return img._pixels[position];
    // }

    // __host__ __device__ void setColor(Image img, size_t x, size_t y, Color3 color)
    // {
    //     int position = y * img._width + x;
    //     img._pixels[position] = color;
    // }

    // __host__ __device__ Ray3 makeRay(Camera cam, size_t width, size_t height, size_t x, size_t y)
    // {
    //     double xR = ((x / (double)width) * 2) - 1;
    //     double yR = ((y / (double)height) * 2) - 1;
    //     auto vec = cam._forward + xR * cam._w * cam._right;
    //     auto direction = cam._forward + xR * cam._w * cam._right;
    //     direction = direction + yR * cam._h * cam._up;
    //     direction = normalize(direction);
    //     return S_ray_new(cam._pos, direction);
    // }

    __host__ __device__ Vector3 calculateRayPoint(Ray3 ray, double t)
    {
        return ray._o + ray._d * t;
    }

    __host__ __device__ Intersection3 intersectShapes(CUDAScene scene, Ray3 ray)
    {
        Intersection3 hit;
        hit.hit = false;
        hit.t = __DBL_MAX__;
        for (int i = 0; i < scene.numShapes; i++)
        {
            auto temp = intersectShape(scene._shapes[i], ray);
            if (temp.t < hit.t)
                hit = temp;
        }
        return hit;
    }

    // __host__ __device__ Intersection3 intersectShapes(thrust::host_vector<TaggedShape> shapes, Ray3 ray)
    // {
    //     Intersection3 hit;
    //     hit.hit = false;
    //     hit.t = __DBL_MAX__;
    //     for (auto shape : shapes)
    //     {
    //         auto temp = intersectShape(shape, ray);
    //         if (temp.hit && temp.t < hit.t)
    //             hit = temp;
    //     }
    //     return hit;
    // }

}