#include "hip/hip_runtime.h"
#include "RenderKernel.h"
#include "Color.hpp"
#include "Scene.hpp"
#include "Camera.hpp"
#include "Ray.hpp"
#include "Ray2.hpp"
#include "Light.hpp"
#include <utility>
#include <random>
#include "Material.hpp"
namespace raytracer
{
    __global__ void renderPar()
    {
    }

    Color renderPixel(int width, int height, int x, int y, const Scene &scene, int bounces, int samples)
    {
        Color pixel{0, 0, 0};
        double reflectivity{0};
        auto ray = scene.camera().makeRay(width, height, Vector2{x, y});
        if (auto hit = scene.shapes().intersect(ray))
        {
            pixel = *hit->color(scene.visibleLights(hit->position()));
            //pixel += globalLight(*hit, scene, samples);
            ray = hit->reflectionRay();
            reflectivity = hit->material()->specularity();
        }
        for (size_t bounce = 1; bounce < bounces; bounce++)
        {
            if (auto hit = scene.shapes().intersect(ray))
            {
                pixel += *hit->color(scene.visibleLights(hit->position())) * reflectivity * (1 / std::pow(2, bounce));
                ray = hit->reflectionRay();
            }
            else
                break;
        }
        return pixel;
    }

    Color globalLight(Intersection &hit, const Scene &scene, int samples)
    {
        Color average{0, 0, 0};
        auto position{hit.position()};
        std::vector<Ray> sampleRays;
        for (int i = 0; i < samples; i++)
        {
            Vector noisyNormal = hit.normal().addNoise();
            Ray noisyRay{hit.position() + 0.1 * hit.normal(), noisyNormal};
            sampleRays.push_back(noisyRay);
        }
        for (auto ray : sampleRays)
        {
            if (auto hit = scene.shapes().intersect(ray))
            {
                double reflectivity{hit->material()->specularity()};
                double distance{hit->t()};
                average += *hit->color(scene.visibleLights(hit->position())) * std::min((reflectivity / distance), (double)1);
            }
        }
        average = average * (1 / (float)samples);
        return average;
    }
}
