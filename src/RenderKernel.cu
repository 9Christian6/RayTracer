#include "hip/hip_runtime.h"
#include "RenderKernel.h"
#include "Color.hpp"
#include "Scene.hpp"
#include "Camera.hpp"
#include "Ray.hpp"
#include "Ray2.hpp"
#include "Light.hpp"
#include <utility>
#include "Material.hpp"
namespace raytracer
{
    __global__ void renderPar()
    {
    }

    Color renderPixel(int width, int height, int x, int y, const Scene &scene, int bounces)
    {
        Color pixel{0, 0, 0};
        auto ray = scene.camera().makeRay(width, height, Vector2{x, y});
        if (auto hit = scene.shapes().intersect(ray))
        {
            pixel = *hit->color(scene.visibleLights(hit->position()));
            ray = hit->reflectionRay();
        }
        for (size_t bounce = 1; bounce < bounces; bounce++)
        {
            if (auto hit = scene.shapes().intersect(ray))
            {
                pixel += *hit->color(scene.visibleLights(hit->position())) * hit.value().material()->specularity() * (1 / (double)bounce);
                ray = hit->reflectionRay();
            }
            else
                break;
        }
        return pixel;
    }

    Color globalLight(Intersection &hit, const Scene &scene, int samples)
    {
        auto position{hit.position()};
        return Color{0, 0, 0};
    }
}