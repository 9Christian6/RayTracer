#include "hip/hip_runtime.h"
#include "RenderKernel.h"
#include "Color.hpp"
#include "Scene.hpp"
#include "Camera.hpp"
#include "Ray.hpp"
#include "Ray2.hpp"
#include "Light.hpp"
#include <utility>
#include <random>
#include "Material.hpp"
namespace raytracer
{
    Color renderPixel(int width, int height, int x, int y, const Scene &scene, int bounces, int samples)
    {
        Color pixel{0, 0, 0};
        double reflectivity{0};
        auto ray = scene.camera().makeRay(width, height, Vector2{x, y});
        if (auto hit = scene.shapes().intersect(ray))
        {
            pixel = *hit->color(scene.visibleLights(hit->position()));
            // pixel += globalLight(*hit, scene, samples);
            ray = hit->reflectionRay();
            reflectivity = hit->material()->specularity();
        }
        for (size_t bounce = 1; bounce < bounces; bounce++)
        {
            if (auto hit = scene.shapes().intersect(ray))
            {
                pixel += *hit->color(scene.visibleLights(hit->position())) * reflectivity * (1 / std::pow(2, bounce));
                ray = hit->reflectionRay();
            }
            else
                break;
        }
        return pixel;
    }

    Color globalLight(Intersection &hit, const Scene &scene, int samples)
    {
        Color average{0, 0, 0};
        auto position{hit.position()};
        std::vector<Ray> sampleRays;
        for (int i = 0; i < samples; i++)
        {
            Vector noisyNormal = hit.normal().addNoise();
            Ray noisyRay{hit.position() + 0.1 * hit.normal(), noisyNormal};
            sampleRays.push_back(noisyRay);
        }
        for (auto ray : sampleRays)
        {
            if (auto hit = scene.shapes().intersect(ray))
            {
                double reflectivity{hit->material()->specularity()};
                double distance{hit->t()};
                average += *hit->color(scene.visibleLights(hit->position())) * std::min((reflectivity / distance), (double)1);
            }
        }
        average = average * (1 / (float)samples);
        return average;
    }

    struct vector3
    {
        int x, y, z;
    };

    __global__ void incVec(vector3 *vec)
    {
        vec->x++;
        vec->y++;
        vec->z++;
    }

    __global__ void vecAdd(double *a, double *b, double *c, int n)
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;

        if (id < n)
            c[id] = a[id] + b[id];
    }

    void incVec3(Vector &vec)
    {
        vector3 vec3;
        vec3.x = vec.x();
        vec3.y = vec.y();
        vec3.z = vec.z();
        vector3 *h_vec, *d_vec;
        size_t bytes = sizeof(vector3);
        h_vec = (vector3 *)malloc(bytes);
        hipMalloc(&d_vec, bytes);
        h_vec->x = vec3.x;
        h_vec->y = vec3.y;
        h_vec->z = vec3.z;
        hipMemcpy(d_vec, h_vec, bytes, hipMemcpyHostToDevice);
        int blockSize = 1024;
        int gridSize = (int)ceil((float)3 / blockSize);
        incVec<<<gridSize, blockSize>>>(d_vec);
        hipMemcpy(h_vec, d_vec, bytes, hipMemcpyDeviceToHost);
        vec = Vector{h_vec->x, h_vec->y, h_vec->z};
        hipFree(d_vec);
        free(h_vec);
    }

    void parTest(Vector &lhs, Vector &rhs)
    {
        double *h_lhs, *h_rhs, *h_result, *d_lhs, *d_rhs, *d_result;

        size_t bytes = 3 * sizeof(double);

        h_lhs = (double *)malloc(bytes);
        h_rhs = (double *)malloc(bytes);
        h_result = (double *)malloc(bytes);

        hipMalloc(&d_lhs, bytes);
        hipMalloc(&d_rhs, bytes);
        hipMalloc(&d_result, bytes);

        h_lhs[0] = lhs.x();
        h_lhs[1] = lhs.y();
        h_lhs[2] = lhs.z();
        h_rhs[0] = rhs.x();
        h_rhs[1] = rhs.y();
        h_rhs[2] = rhs.z();

        hipMemcpy(d_lhs, h_lhs, bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_rhs, h_rhs, bytes, hipMemcpyHostToDevice);

        int blockSize = 1024;
        int gridSize = (int)ceil((float)3 / blockSize);

        vecAdd<<<gridSize, blockSize>>>(d_lhs, d_rhs, d_result, 3);

        hipMemcpy(h_result, d_result, bytes, hipMemcpyDeviceToHost);

        for (int i = 0; i < 3; i++)
        {
            std::cout << h_result[i] << "\n";
        }

        hipFree(d_lhs);
        hipFree(d_rhs);
        hipFree(d_result);

        free(h_lhs);
        free(h_rhs);
        free(h_result);
    }
}
