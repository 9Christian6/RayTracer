#include "hip/hip_runtime.h"
#include "CUDA.h"
#include <iostream>

__global__ void vector_add(double *result, const double *lhs, const double *rhs)
{
    *result = 0;
}

void test(double *asdf)
{
    double *d;
    hipMalloc(&d, sizeof(*d));
    hipMemcpy(d, asdf, sizeof(double), hipMemcpyHostToDevice);
    vector_add<<<1, 10>>>(d, nullptr, nullptr);
    hipMemcpy(asdf, d, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d);
}